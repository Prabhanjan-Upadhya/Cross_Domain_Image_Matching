#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#include<time.h>
#include<sys/time.h>

#define SQUARE(x) ((x)*(x))
#define PI 3.14
#define BLOCK_SIZE 16
#define N 4096

__device__ __constant__ float pi=3.14;
__device__ __constant__ int sobelX[9]={1,0,-1,2,0,-2,1,0,-1};
__device__ __constant__ int sobelY[9]={1,2,1,0,0,0,-1,-2,-1};


__global__ void NormalizeGrayGPU(double input[], int width, int height, unsigned char output[], double min, double max)
{
	int index= blockIdx.x*blockDim.x+threadIdx.x;
	
	if(index < (width*height))
	{
		output[index]=(input[index]-min)*255/(max-min);
	}
}

double FindMin(double input[], int width, int height)
{
	double min = input[0];
	
	for (int i = 0; i < width*height; i++)
	{
		if (input[i] < min) min = input[i];
	}
	return min;
}
double FindMax(double input[], int width, int height)
{
	double max = input[0];
	
	for (int i = 0; i < width*height; i++)
	{
		if (input[i] > max) max = input[i];
	}
	return max;
}

__global__ void SobelFilter_gpu(unsigned char* A, double *gradImageX, double *gradImageY, double *gradMag, int width, int height)
{
	

	int row= blockIdx.y*blockDim.y+threadIdx.y;
	int col= blockIdx.x*blockDim.x+threadIdx.x;
	double tempx=0;
	double tempy=0;	
		if(row < height && col < width){


			tempx = 0;
			tempy = 0;
			for (int r2=-1; r2<=1; r2++){
				for (int c2=-1; c2<=1; c2++)
				{	
					tempx += A[(row+r2)*width+(col+c2)]*sobelX[(r2+1)*3+c2+1];
					tempy += A[(row+r2)*width+(col+c2)]*sobelY[(r2+1)*3+c2+1];
				}
			}


			gradImageX[(row*width)+col]=tempx;
			gradImageY[(row*width)+col]=tempy;
			gradMag[(row*width)+col]= sqrt((double) (tempx*tempx)+(tempy*tempy));

	
		}
		
}
		


__global__ void theta_gpu(double *gradImageY, double *gradImageX, double *gradPhase, int width, int height){
	
	int index= blockIdx.x*blockDim.x+threadIdx.x;
	if(index<(width*height)){
		float theta = atan2(gradImageY[index],gradImageX[index]);
		theta=theta*180/pi;
		gradPhase[index]=theta;
	}

}




int main(int argc, char *argv[])
{

	FILE *fptr;
	char *inputHeader, *testHeader;
	int inputCols, inputRows, inputBytes;
	int testCols, testRows, testBytes;
	char Header_1[320], Header_2[320];
	unsigned char *inputImage, *testImage;
	unsigned char *normalGradMag, *normalGrad_x, *normalGrad_y, *normalGradPhase;
	unsigned char *normaltestMag, *normaltest_x, *normaltest_y, *normaltestPhase;
	double *gradPhase, *gradMag;
	double *testgradPhase, *testgradMag;
	double max=0;
	double min=0;
	float gpu_time_1 = 0;
	float gpu_time_2 = 0;
	float gpu_time_3 = 0;

	//GPU variables
	double *d_gradImageX, *d_gradImageY, *d_gradPhase, *d_gradMag;
	unsigned char *d_inputImage, *d_normalGradMag, *d_normalGradX, *d_normalGradY, *d_normalGradPhase;
	unsigned char *d_testImage;
	double *d_testgradImageX, *d_testgradImageY, *d_testgradMag, *d_testgradPhase;
	unsigned char *d_testnormalGradMag, *d_testnormalGradX, *d_testnormalGradY, *d_testnormalGradPhase;
	hipError_t err;
	struct timeval cstart1, cstart2, cstart3, cend1, cend2, cend3;
	hipEvent_t start1, start2, start3, stop1, stop2, stop3;
	
	printf("Initialization done!\n");
	
	gettimeofday(&cstart1, NULL);
	if ((fptr=fopen(argv[1],"r"))==NULL)
	{
		printf("Unable to open input file for reading\n");
		exit(0);
	}

	//Open and load input image
	fptr = fopen(argv[1], "r");
	fscanf(fptr,"%s %d %d %d",&inputHeader, &inputCols, &inputRows, &inputBytes);	
	Header_1[0]=fgetc(fptr);	/* read white-space character that separates header */
	inputImage = (unsigned char*)calloc(inputCols*inputRows,sizeof(unsigned char));
	fread(inputImage, 1, inputCols*inputRows, fptr);
	fclose(fptr);
	printf("Input file opened!\n");

	if ((fptr = fopen(argv[2], "r")) == NULL)
	{
		printf("Unable to open test file for reading\n");
		exit(0);
	}
	//Open and load test image
	fptr = fopen(argv[2], "rb");
	fscanf(fptr, "%s %d %d %d", &testHeader, &testCols, &testRows, &testBytes);
	Header_2[0] = fgetc(fptr);	/* read white-space character that separates header */
	testImage = (unsigned char*)calloc(testCols*testRows, sizeof(unsigned char));
	fread(testImage, 1, testCols*testRows, fptr);
	fclose(fptr);
	printf("Test file opened!\n");

	gettimeofday(&cend1, NULL);
	

	hipEventCreate(&start1);
	hipEventCreate(&stop1);

	hipEventRecord(start1);
	hipEventSynchronize(start1);

	//cudaMalloc for Input image
	err=hipMalloc(&d_inputImage,(inputRows*inputCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_inputImage");

	err=hipMalloc(&d_gradImageX,(inputRows*inputCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_gradImageX");

	err=hipMalloc(&d_gradImageY,(inputRows*inputCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_gradImageY");

	err=hipMalloc(&d_gradPhase,(inputRows*inputCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_gradPhase");

	err=hipMalloc(&d_gradMag,(inputRows*inputCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_gradMag");

	err=hipMalloc(&d_normalGradMag,(inputRows*inputCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_normalGradMag");

	err=hipMalloc(&d_normalGradX,(inputRows*inputCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_normalGradX");
	
	err=hipMalloc(&d_normalGradY,(inputRows*inputCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_normalGradY");
	
	err=hipMalloc(&d_normalGradPhase,(inputRows*inputCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_normalGradPhase");

	//cudaMalloc for test image
	err=hipMalloc(&d_testImage,(testRows*testCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testImage");

	err=hipMalloc(&d_testgradImageX,(testRows*testCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testgradImageX");

	err=hipMalloc(&d_testgradImageY,(testRows*testCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testgradImageY");

	err=hipMalloc(&d_testgradPhase,(testRows*testCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testgradPhase");

	err=hipMalloc(&d_testgradMag,(testRows*testCols*sizeof(double)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testgradMag");

	err=hipMalloc(&d_testnormalGradMag,(testRows*testCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testnormalGradMag");

	err=hipMalloc(&d_testnormalGradX,(testRows*testCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testnormalGradX");
	
	err=hipMalloc(&d_testnormalGradY,(testRows*testCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testnormalGradY");
	
	err=hipMalloc(&d_testnormalGradPhase,(testRows*testCols*sizeof(unsigned char)));
	if(err != hipSuccess) printf("/n Error in hipMalloc d_testnormalGradPhase");
	
	hipEventSynchronize(stop1);
	hipEventRecord(stop1);

	gettimeofday(&cstart2, NULL);
	//Normalized input gradient images
	normalGradMag = (unsigned char*)calloc(inputCols*inputRows, sizeof(unsigned char));
	normalGrad_x = (unsigned char*)calloc(inputCols*inputRows, sizeof(unsigned char));
	normalGrad_y = (unsigned char*)calloc(inputCols*inputRows, sizeof(unsigned char));
	normalGradPhase = (unsigned char*)calloc(inputCols*inputRows, sizeof(unsigned char));
	gradPhase = (double*)calloc(inputCols*inputRows, sizeof(double));
	gradMag = (double*)calloc(inputCols*inputRows, sizeof(double));
	
	//Normalized test gradient images
	normaltestMag = (unsigned char*)calloc(testCols*testRows, sizeof(unsigned char));
	normaltest_x = (unsigned char*)calloc(testCols*testRows, sizeof(unsigned char));
	normaltest_y = (unsigned char*)calloc(testCols*testRows, sizeof(unsigned char));
	normaltestPhase = (unsigned char*)calloc(testCols*testRows, sizeof(unsigned char));
	testgradPhase = (double*)calloc(testCols*testRows, sizeof(double));
	testgradMag = (double*)calloc(testCols*testRows, sizeof(double));
	
	gettimeofday(&cend2, NULL);	
	
	hipEventCreate(&start2);
	hipEventCreate(&stop2);

	hipEventRecord(start2);
	hipEventSynchronize(start2);
	//Compute gradients and phase for input image
	err=hipMemcpy(d_inputImage, inputImage, (inputRows*inputCols*sizeof(unsigned char)), hipMemcpyHostToDevice);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of d_inputImage");

	/* Launch Kernel*/
	 dim3 dimGrid(ceil((float)(N+2)/BLOCK_SIZE), ceil((float)(N+2)/BLOCK_SIZE),1);
	 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	
	SobelFilter_gpu<<<dimGrid,dimBlock>>>(d_inputImage, d_gradImageX, d_gradImageY, d_gradMag, inputCols, inputRows);
	hipDeviceSynchronize();

	dim3 BlockDim = dim3(1024,1,1);
	dim3 GridDim = dim3(10000,1,1);
	theta_gpu<<<GridDim,BlockDim>>>(d_gradImageY,d_gradImageX, d_gradPhase, inputCols, inputRows);


	//Compute gradients and phase for test image
	err=hipMemcpy(d_testImage, testImage, (testRows*testCols*sizeof(unsigned char)), hipMemcpyHostToDevice);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of d_testImage");

	/* Launch Kernel*/
	SobelFilter_gpu<<<dimGrid,dimBlock>>>(d_testImage, d_testgradImageX, d_testgradImageY, d_testgradMag, testCols, testRows);
	hipDeviceSynchronize();
	theta_gpu<<<GridDim,BlockDim>>>(d_testgradImageY,d_testgradImageX, d_testgradPhase, testCols, testRows);
	
	hipMemcpy(gradMag, d_gradMag,(inputCols*inputRows*sizeof(double)),hipMemcpyDeviceToHost);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of normalGrad_x");
	min = FindMin(gradMag, inputCols, inputRows);
	max = FindMax(gradMag, inputCols, inputRows);
	NormalizeGrayGPU<<<GridDim,BlockDim>>>(d_gradMag, inputCols, inputRows, d_normalGradMag, min, max);
	hipDeviceSynchronize();
	
	hipMemcpy(testgradMag, d_testgradMag,(inputCols*inputRows*sizeof(double)),hipMemcpyDeviceToHost);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of normalGrad_x");
	min = FindMin(testgradMag, testCols, testRows);
	max = FindMax(testgradMag, testCols, testRows);
	NormalizeGrayGPU<<<GridDim,BlockDim>>>(d_testgradMag, testCols, testRows, d_testnormalGradMag, min, max);
	hipDeviceSynchronize();

	hipMemcpy(gradPhase, d_gradPhase,(inputCols*inputRows*sizeof(double)),hipMemcpyDeviceToHost);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of gradPhase");
	
	hipMemcpy(testgradPhase, d_testgradPhase,(testCols*testRows*sizeof(double)),hipMemcpyDeviceToHost);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of testgradPhase");
	
	hipMemcpy(normalGradMag, d_normalGradMag,(inputCols*inputRows*sizeof(unsigned char)),hipMemcpyDeviceToHost);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of normalGradMag");
	
	hipMemcpy(normaltestMag, d_testnormalGradMag,(testCols*testRows*sizeof(unsigned char)),hipMemcpyDeviceToHost);
	if(err != hipSuccess) printf("/n Error in hipMemcpy of normaltestMag");
	
	hipEventRecord(stop2);
	hipEventSynchronize(stop2);
	
	gettimeofday(&cstart3, NULL);	
	int histo[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int testhisto[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int difference[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	//Compute histogram of gradient orientations of input image
	double angle = 0;
	for (int i = 0; i < inputRows*inputCols; i++)
	{
		if (normalGradMag[i] > 25)
		{
			angle = fabs(gradPhase[i]);
			if (angle > 0 && angle < 21) histo[0]++;
			else if (angle > 21 && angle < 41) histo[1]++;
			else if (angle > 41 && angle < 61) histo[2]++;
			else if (angle > 61 && angle < 81) histo[3]++;
			else if (angle > 81 && angle < 101) histo[4]++;
			else if (angle > 101 && angle < 121) histo[5]++;
			else if (angle > 121 && angle < 141) histo[6]++;
			else if (angle > 141 && angle < 161) histo[7]++;
			else histo[8]++;
		}
	}

	printf("here6\n");
	//Compute histogram of gradient orientations of test image
	angle = 0;
	for (int i = 0; i < testRows*testCols; i++)
	{
		if (normaltestMag[i] > 25)
		{
			angle = fabs(testgradPhase[i]);
			if (angle > 0 && angle < 21) testhisto[0]++;
			else if (angle > 21 && angle < 41) testhisto[1]++;
			else if (angle > 41 && angle < 61) testhisto[2]++;
			else if (angle > 61 && angle < 81) testhisto[3]++;
			else if (angle > 81 && angle < 101) testhisto[4]++;
			else if (angle > 101 && angle < 121) testhisto[5]++;
			else if (angle > 121 && angle < 141) testhisto[6]++;
			else if (angle > 141 && angle < 161) testhisto[7]++;
			else testhisto[8]++;
		}
	}

	printf("here7\n");
	//Check the dissimilarity in histogram of gradient orientations
	int sumDiff = 0;
	for (int i = 0; i < 9; i++)
	{
		difference[i] = abs(histo[i] - testhisto[i]);
		printf("diff[%d] = %d\n", i, difference[i]);
		sumDiff += difference[i];
	}
	//float mismatch = (float)sumDiff*100/(testCols*testRows);
	printf("HOG mismatch = %d\n", sumDiff);
	

	fptr=fopen("input_grad_mag.pgm","w");
	fprintf(fptr,"P5 %d %d 255\n",inputCols,inputRows);
	fwrite(normalGradMag,inputCols*inputRows,1,fptr);
	fclose(fptr);

	fptr=fopen("test_grad_mag.pgm","w");
	fprintf(fptr,"P5 %d %d 255\n",testCols,testRows);
	fwrite(normaltestMag,testCols*testRows,1,fptr);
	fclose(fptr);
	
	//Free allocated memory
	free(normalGradMag);
	free(normalGradPhase);
	free(normalGrad_x);
	free(normalGrad_y);
	free(normaltestMag);
	free(normaltestPhase);
	free(normaltest_x);
	free(normaltest_y);
	
	gettimeofday(&cend3, NULL);
	
	//Free Allocated memory on the device. Don't forget. 
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	hipEventRecord(start3);
	hipEventSynchronize(start3);

	hipFree(d_gradImageX);
	hipFree(d_gradImageY);
	hipFree(d_gradPhase);
	hipFree(d_gradMag);
	hipFree(d_inputImage);
	hipFree(d_normalGradMag);
	hipFree(d_normalGradX);
	hipFree(d_normalGradY);
	hipFree(d_normalGradPhase);	

	hipFree(d_testgradImageX);
	hipFree(d_testgradImageY);
	hipFree(d_testgradPhase);
	hipFree(d_testgradMag);
	hipFree(d_testImage);
	hipFree(d_testnormalGradMag);
	hipFree(d_testnormalGradX);
	hipFree(d_testnormalGradY);
	hipFree(d_testnormalGradPhase);
	
	hipEventRecord(stop3);
	hipEventSynchronize(stop3);
	
	//Calculate time tiaken
//	float gpu_time_1 = 0;
//	float gpu_time_2 = 0;
//	float gpu_time_3 = 0;
	hipEventElapsedTime(&gpu_time_1, start1, stop1);
	hipEventElapsedTime(&gpu_time_2, start2, stop2);
	hipEventElapsedTime(&gpu_time_3, start3, stop3);
	printf("gpu_time_1 = %f\t gpu_time_2 = %f\t gpu_time_3 = %f\n",gpu_time_1, gpu_time_2, gpu_time_3);
	printf("Total GPU time = %f\n", gpu_time_1+gpu_time_2+gpu_time_3);
	
	float cpu_time_1 = (((cend1.tv_sec * 1000000 + cend1.tv_usec) - (cstart1.tv_sec * 1000000 + cstart1.tv_usec))/1000.0);
	float cpu_time_2 = (((cend2.tv_sec * 1000000 + cend2.tv_usec) - (cstart2.tv_sec * 1000000 + cstart2.tv_usec))/1000.0);
	float cpu_time_3 = (((cend3.tv_sec * 1000000 + cend3.tv_usec) - (cstart3.tv_sec * 1000000 + cstart3.tv_usec))/1000.0);

	printf("cpu_time_1 = %f\t cpu_time_2 = %f\t cpu_time_3 = %f\n",cpu_time_1, cpu_time_2, cpu_time_3);
	printf("Total CPU time = %f\n", cpu_time_1+cpu_time_2+cpu_time_3);

	printf(" Total time = %f\n", gpu_time_1+gpu_time_2+gpu_time_3+ cpu_time_1+cpu_time_2+cpu_time_3);
	
	return 0;
}
